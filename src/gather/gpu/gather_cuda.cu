
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cstdio>
using ull = unsigned long long;

static int indices_size, inputR_stride, axis_size, output_size;
constexpr int block_size = 128;

template<typename T>
__global__ void gather(const T *input, const ull*indices, T *output, int N, int axis_size, int inputR_stride)
{
    /* 
        gridDim.x : indices_size
        gridDim.y : inputL
        blockIdx.x : indices
        blockIdx.y : inputL
        threadIdx.x : inputR
    */
    int output_offset = blockIdx.x * inputR_stride + blockIdx.y * gridDim.x * inputR_stride;
    int input_offset = blockIdx.y * axis_size * inputR_stride + indices[blockIdx.x] * inputR_stride;
    for(int tid = threadIdx.x; tid < inputR_stride; tid += block_size)
        if(output_offset + tid < N)
            output[output_offset + tid] = input[input_offset + tid];
}

template<typename T>
void gatherLaunch(const void *input, const void *indices, void *output)
{
    int base = indices_size * inputR_stride;
    int inputL = (output_size + base - 1) / base;
    dim3 grid_size(indices_size, inputL);

    gather<T><<<grid_size, block_size>>>(static_cast<const T*>(input), 
                                         static_cast<const ull*>(indices), 
                                         static_cast<T*>(output), 
                                         output_size, axis_size, inputR_stride);
}

static void dataPreprocess(const int *input_shape, const int input_shape_len, 
                           const int *indices_shape, const int indices_shape_len, 
                           const int axis)
{
    axis_size = input_shape[axis];
    indices_size = inputR_stride = output_size = 1;
    for(int i = 0; i < indices_shape_len; i++)
        indices_size *= indices_shape[i];
    
    for(int i = input_shape_len - 1; i >= 0; i--)
    {
        output_size *= input_shape[i];
        if(i > axis)
            inputR_stride *= input_shape[i];
    }
    output_size = output_size / axis_size * indices_size;
}

extern "C" void gather_nv_f32(const void *input, const void *indices, void *output, const int axis, 
                              const int *input_shape, const int input_shape_len, 
                              const int *indices_shape, const int indices_shape_len)
{
    dataPreprocess(input_shape, input_shape_len, indices_shape, indices_shape_len, axis);
    gatherLaunch<float>(input, indices, output);
}

extern "C" void gather_nv_f16(const void *input, const void *indices, void *output, const int axis, 
                              const int *input_shape, const int input_shape_len, 
                              const int *indices_shape, const int indices_shape_len)
{
    dataPreprocess(input_shape, input_shape_len, indices_shape, indices_shape_len, axis);
    gatherLaunch<half>(input, indices, output);
}
